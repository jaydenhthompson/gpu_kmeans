#include "hip/hip_runtime.h"
#include "helpers.h"
#include "kmeans.h"

// CUDA Includes
#include <hip/hip_runtime.h>

// thrust includes
#include <thrust/device_vector.h>

#include <iostream>
#include <vector>

__device__ double euclideanDistance(double *a, double *b, int dim)
{
    double dist = 0.0;
    for(int i = 0; i < dim; i++)
    {
        dist += pow(a[i] - b[i], 2);
    }
    return sqrt(dist);
}

__global__ void cudaCalculateFlags(double *d_data, double *d_centroids, int *d_flags, int *d_numAssigned, int d_dataSize, int d_numCentroids, int d_dimensions)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(index >= d_dataSize) return;

    int assigned = 0;
    double minDist = INFINITY;
    for (int i = 0; i < d_numCentroids; i++)
    {
        double dist = euclideanDistance(&d_data[index*d_dimensions], &d_centroids[i*d_dimensions], d_dimensions);
        if(dist < minDist)
        {
            minDist = dist;
            assigned = i;
        }
    }
    atomicAdd(&d_numAssigned[assigned], 1);
    d_flags[index] = assigned;
}

__global__ void cudaAddNewCentroids(double *d_data, double *d_centroids, int *d_flags, int *d_numAssigned, int d_dataSize, int d_numCentroids, int d_dimensions)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= d_dataSize) return;

    int assignedCentroid = d_flags[index];

    for(int i = 0; i < d_dimensions; i++)
    {
        atomicAdd(&d_centroids[assignedCentroid*d_dimensions + i], d_data[index*d_dimensions + i]);
    }
}

__global__ void cudaAverageNewCentroids(double *d_data, double *d_centroids, int *d_flags, int *d_numAssigned, int d_dataSize, int d_numCentroids, int d_dimensions)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= d_dataSize) return;

    if(index < d_numCentroids)
    {
        for(int i = 0; i < d_dimensions; i++)
        {
            d_centroids[index*d_dimensions + i] /= d_numAssigned[index];
        }
    }
}

__global__ void shmemCalculateFlags(double *d_data, double *d_centroids, int *d_flags, int *d_numAssigned, int d_dataSize, int d_numCentroids, int d_dimensions)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int sharedIndex = threadIdx.x;

    if (index >= d_dataSize)
        return;

    extern __shared__ int s_numAssigned[];
    if (sharedIndex == 0)
    {
        for (int i = 0; i < d_numCentroids; i++)
        {
            s_numAssigned[i] = 0;
        }
    }
    __syncthreads();

    int assigned = 0;
    double minDist = INFINITY;
    for (int i = 0; i < d_numCentroids; i++)
    {
        double dist = euclideanDistance(&d_data[index * d_dimensions], &d_centroids[i * d_dimensions], d_dimensions);
        if (dist < minDist)
        {
            minDist = dist;
            assigned = i;
        }
    }
    d_flags[index] = assigned;
    atomicAdd(&s_numAssigned[assigned], 1);

    __syncthreads();
    if (sharedIndex == 0)
    {
        for (int i = 0; i < d_numCentroids; i++)
        {
            atomicAdd(&d_numAssigned[i], s_numAssigned[i]);
        }
    }
}

__global__ void shmemAddNewCentroids(double *d_data, double *d_centroids, int *d_flags, int *d_numAssigned, int d_dataSize, int d_numCentroids, int d_dimensions)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= d_dataSize) return;

    int assignedCentroid = d_flags[index];

    for(int i = 0; i < d_dimensions; i++)
    {
        atomicAdd(&d_centroids[assignedCentroid*d_dimensions + i], d_data[index*d_dimensions + i]);
    }
}

__global__ void shmemAverageNewCentroids(double *d_data, double *d_centroids, int *d_flags, int *d_numAssigned, int d_dataSize, int d_numCentroids, int d_dimensions)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < d_numCentroids)
    {
        for (int i = 0; i < d_dimensions; i++)
        {
            d_centroids[index * d_dimensions + i] /= d_numAssigned[index];
        }
    }
}

std::vector<float> runCuda(const std::vector<double> &data, std::vector<double> &centroids, std::vector<int> &flags, int option, int dimensions, int numData, int numClusters, int maxIterations, double threshold)
{
    ////////////////////
    // Host variables //
    ////////////////////

    std::vector<double> newCentroids(centroids.size(), 0);
    std::vector<float> times;

    ////////////////////
    // CUDA variables //
    ////////////////////

    double *d_data = nullptr;
    int dataSize = data.size() * sizeof(double);
    hipMalloc((void**)&d_data, dataSize);
    hipMemcpy(d_data, &data[0], dataSize, hipMemcpyHostToDevice);

    double *d_centroids = nullptr;
    int centroidSize = centroids.size() * sizeof(double);
    hipMalloc((void**)&d_centroids, centroidSize);
    hipMemcpy(d_centroids, &centroids[0], centroidSize, hipMemcpyHostToDevice);

    double *d_newCentroids = nullptr;
    hipMalloc((void**)&d_newCentroids, centroidSize);
    hipMemcpy(d_newCentroids, &centroids[0], centroidSize, hipMemcpyHostToDevice);

    int *d_flags = nullptr;
    int flagsSize = flags.size() * sizeof(int);
    hipMalloc((void**)&d_flags, flagsSize);

    int *d_numAssigned = nullptr;
    hipMalloc((void**)&d_numAssigned, numClusters * sizeof(int));

    ////////////////////
    // Execution Loop //
    ////////////////////

    int threadsPerBlock = 256;
    int blocksPerGrid = (numData + threadsPerBlock - 1) / threadsPerBlock;
    for (int i = 0; i < maxIterations; i++)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        /////////////////
        // Run Kernels //
        /////////////////

        hipMemset(d_numAssigned, 0, numClusters * sizeof(int));
        switch(option)
        {
        case 1:
            cudaCalculateFlags<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_newCentroids, d_flags, d_numAssigned, numData, numClusters, dimensions);
            hipDeviceSynchronize();
            hipMemset(d_newCentroids, 0, centroidSize);
            cudaAddNewCentroids<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_newCentroids, d_flags, d_numAssigned, numData, numClusters, dimensions);
            hipDeviceSynchronize();
            cudaAverageNewCentroids<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_newCentroids, d_flags, d_numAssigned, numData, numClusters, dimensions);
            hipDeviceSynchronize();
            break;
        case 2:
            shmemCalculateFlags<<<blocksPerGrid, threadsPerBlock, numClusters*sizeof(int)>>>(d_data, d_newCentroids, d_flags, d_numAssigned, numData, numClusters, dimensions);
            hipDeviceSynchronize();
            hipMemset(d_newCentroids, 0, centroidSize);
            shmemAddNewCentroids<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_newCentroids, d_flags, d_numAssigned, numData, numClusters, dimensions);
            hipDeviceSynchronize();
            shmemAverageNewCentroids<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_newCentroids, d_flags, d_numAssigned, numData, numClusters, dimensions);
            hipDeviceSynchronize();
            break;
        default:
            std::cout << "Wrong option for -r" << std::endl;
            return times;
            break;
        }

        /////////////////////////
        // Calculate Threshold //
        /////////////////////////

        hipMemcpy(&newCentroids[0], d_newCentroids, centroidSize, hipMemcpyDeviceToHost);
        auto movement = calculateVectorMovement(centroids, newCentroids, numClusters, dimensions);
        /////////////////
        // Record Time //
        /////////////////

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float time = 0;
        hipEventElapsedTime(&time, start, stop);
        times.push_back(time);

        // record new centroids
        centroids = newCentroids;
        if(movement <= threshold)
        {
            break;
        }
    }

    // record flags
    hipMemcpy(flags.data(), d_flags, flagsSize, hipMemcpyDeviceToHost);

    // memory management
    hipFree(d_data);
    hipFree(d_centroids);
    hipFree(d_newCentroids);
    hipFree(d_flags);

    return times;
}

std::vector<float> runThrust(const std::vector<double> &data, std::vector<double> &centroids, std::vector<int> &flags, int option, int dimensions, int numData, int numClusters, int maxIterations, double threshold)
{
    std::vector<double> newCentroids(centroids.size(), 0);
    std::vector<float> times;

    thrust::device_vector<double> d_data(data);
    thrust::device_vector<double> d_centroids(centroids);
    thrust::device_vector<double> d_newCentroids(centroids);
    thrust::device_vector<int> d_flags(flags);
    thrust::device_vector<int> d_numAssigned(centroids.size());

    thrust::device_vector<int> originalCentroidOrder(centroids.size() * dimensions);
    thrust::sequence(originalCentroidOrder.begin(), originalCentroidOrder.end());

    thrust::device_vector<int> originalDataOrder(data.size() * dimensions);
    thrust::sequence(originalDataOrder.begin(), originalDataOrder.end());


    ////////////////////
    // Execution Loop //
    ////////////////////

    int threadsPerBlock = 256;
    int blocksPerGrid = (numData + threadsPerBlock - 1) / threadsPerBlock;
    for (int i = 0; i < maxIterations; i++)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        /////////////////
        // Run Kernels //
        /////////////////

        thrust::fill(d_numAssigned.begin(), d_numAssigned.end(), 0);
        cudaCalculateFlags<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(d_data.data()), thrust::raw_pointer_cast(d_newCentroids.data()), thrust::raw_pointer_cast(d_flags.data()), thrust::raw_pointer_cast(d_numAssigned.data()), numData, numClusters, dimensions);
        hipDeviceSynchronize();
        thrust::fill(d_newCentroids.begin(), d_newCentroids.end(), 0);
        cudaAddNewCentroids<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(d_data.data()), thrust::raw_pointer_cast(d_newCentroids.data()), thrust::raw_pointer_cast(d_flags.data()), thrust::raw_pointer_cast(d_numAssigned.data()), numData, numClusters, dimensions);
        hipDeviceSynchronize();
        cudaAverageNewCentroids<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(d_data.data()), thrust::raw_pointer_cast(d_newCentroids.data()), thrust::raw_pointer_cast(d_flags.data()), thrust::raw_pointer_cast(d_numAssigned.data()), numData, numClusters, dimensions);
        hipDeviceSynchronize();

        /////////////////////////
        // Calculate Threshold //
        /////////////////////////

        thrust::copy(d_newCentroids.begin(), d_newCentroids.end(), newCentroids.begin());
        auto movement = calculateVectorMovement(centroids, newCentroids, numClusters, dimensions);
        /////////////////
        // Record Time //
        /////////////////

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float time = 0;
        hipEventElapsedTime(&time, start, stop);
        times.push_back(time);

        // record new centroids
        centroids = newCentroids;
        if(movement <= threshold)
        {
            break;
        }
    }

    return times;
}