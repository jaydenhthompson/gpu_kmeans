#include "hip/hip_runtime.h"
#include "helpers.h"
#include "kmeans.h"

// CUDA Includes
#include <hip/hip_runtime.h>

#include <vector>

__device__ double euclideanDistance(double *a, double *b, int dim)
{
    double dist = 0.0;
    for(int i = 0; i < dim; i++)
    {
        dist += pow(a[i] - b[i], 2);
    }
    return sqrt(dist);
}

__global__ void cudaCalculateFlags(double *d_data, double *d_centroids, int *d_flags, int *d_numAssigned, int d_dataSize, int d_numCentroids, int d_dimensions)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(index >= d_dataSize) return;

    int assigned = 0;
    double minDist = INFINITY;
    for (int i = 0; i < d_numCentroids; i++)
    {
        double dist = euclideanDistance(&d_data[index*d_dimensions], &d_centroids[i*d_dimensions], d_dimensions);
        if(dist < minDist)
        {
            minDist = dist;
            assigned = i;
        }
    }
    atomicAdd(&d_numAssigned[assigned], 1);
    d_flags[index] = assigned;

    __syncthreads();
}

__global__ void cudaCalculateNewCentroids(double *d_data, double *d_centroids, int *d_flags, int *d_numAssigned, int d_dataSize, int d_numCentroids, int d_dimensions)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= d_dataSize) return;

    int assignedCentroid = d_flags[index];

    for(int i = 0; i < d_dimensions; i++)
    {
        atomicAdd(&d_centroids[assignedCentroid*d_dimensions + i], d_data[index*d_dimensions + i]);
    }

    __syncthreads();

    if(index < d_numCentroids)
    {
        for(int i = 0; i < d_dimensions; i++)
        {
            d_centroids[index*d_dimensions + i] /= d_numAssigned[index];
        }
    }
}


std::vector<float> runCudaBasic(const std::vector<double> &data, std::vector<double> &centroids, std::vector<int> &flags, int dimensions, int numData, int numClusters, int maxIterations, double threshold)
{
    std::vector<double> newCentroids(centroids.size(), 0);
    std::vector<float> times;

    double *d_data = nullptr;
    int dataSize = data.size() * sizeof(double);
    hipMalloc((void**)&d_data, dataSize);
    hipMemcpy(d_data, &data[0], dataSize, hipMemcpyHostToDevice);

    double *d_centroids = nullptr;
    int centroidSize = centroids.size() * sizeof(double);
    hipMalloc((void**)&d_centroids, centroidSize);
    hipMemcpy(d_centroids, &centroids[0], centroidSize, hipMemcpyHostToDevice);

    double *d_newCentroids = nullptr;
    hipMalloc((void**)&d_newCentroids, centroidSize);
    hipMemset(d_newCentroids, 0, centroidSize);

    int *d_flags = nullptr;
    int flagsSize = flags.size() * sizeof(int);
    hipMalloc((void**)&d_flags, flagsSize);

    int *d_numAssigned = nullptr;
    hipMalloc((void**)&d_numAssigned, numClusters * sizeof(int));

    int threadsPerBlock = 256;
    int blocksPerGrid = (numData + threadsPerBlock - 1) / threadsPerBlock;
    for (int i = 0; i < maxIterations; i++)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        hipMemset(d_numAssigned, 0, numClusters * sizeof(int));
        cudaCalculateFlags<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_centroids, d_flags, d_numAssigned, numData, numClusters, dimensions);
        cudaCalculateNewCentroids<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_newCentroids, d_flags, d_numAssigned, numData, numClusters, dimensions);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipMemcpy(&newCentroids[0], d_newCentroids, centroidSize, hipMemcpyDeviceToHost);

        auto movement = calculateVectorMovement(centroids, newCentroids, numClusters, dimensions);
        if(movement <= threshold)
        {
            break;
        }

        float time = 0;
        hipEventElapsedTime(&time, start, stop);
        times.push_back(time);

        
        centroids = newCentroids;
        hipMemset(d_newCentroids, 0, centroidSize);
    }

    hipMemcpy(flags.data(), d_flags, flagsSize, hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipFree(d_centroids);
    hipFree(d_newCentroids);
    hipFree(d_flags);
    return times;
}

std::vector<float> runCudaShmem(const std::vector<double> &data, std::vector<double> &centroids, std::vector<int> &flags, int maxIterations, double threshold)
{
    std::vector<float>times;
    return times;
}

std::vector<float> runThrust(const std::vector<double> &data, std::vector<double> &centroids, std::vector<int> &flags, int maxIterations, double threshold)
{
    std::vector<float>times;
    return times;
}
